
#include <hip/hip_runtime.h>
#include <stdio.h>
using namespace std;

const int THREADS_NUM = 1000000;
const int ARRAY_SIZE = 10;
const int BLOCK_WIDTH = 1000;

__global__ void imcrement_naive(int *arr)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    idx = idx % ARRAY_SIZE;

    arr[idx] = arr[idx] + 1;
}

__global__ void imcrement_atomic(int *arr)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    idx = idx % ARRAY_SIZE;

    atomicAdd(&arr[idx], 1);
}

int main()
{
    int h_arr[ARRAY_SIZE] = {0};
    int *d_arr;

    hipMalloc((void **)&d_arr, sizeof(int) * ARRAY_SIZE);
    hipMemset((void *)d_arr, 0, ARRAY_SIZE * sizeof(int));

    hipMemcpy(d_arr, h_arr, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);

    imcrement_naive<<<THREADS_NUM / BLOCK_WIDTH, BLOCK_WIDTH>>>(d_arr);

    hipEventRecord(end);

    hipMemcpy(h_arr, d_arr, ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++)
    {
        printf("%d \t", h_arr[i]);
        if ((i + 1) % 5 == 0)
            printf("\n");
    }
    printf("\n");
    float miliseconds = 0.0;
    hipEventElapsedTime(&miliseconds, start, end);

    printf("CUDA: Kernel execution time: %f ms\n\n", miliseconds);

    hipFree(d_arr);
    return 0;
}