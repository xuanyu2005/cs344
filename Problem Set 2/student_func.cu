#include "hip/hip_runtime.h"
// Homework 2
// Image Blurring
//
// In this homework we are blurring an image. To do this, imagine that we have
// a square array of weight values. For each pixel in the image, imagine that we
// overlay this square array of weights on top of the image such that the center
// of the weight array is aligned with the current pixel. To compute a blurred
// pixel value, we multiply each pair of numbers that line up. In other words, we
// multiply each weight with the pixel underneath it. Finally, we add up all of the
// multiplied numbers and assign that value to our output for the current pixel.
// We repeat this process for all the pixels in the image.

// To help get you started, we have included some useful notes here.

//****************************************************************************

// For a color image that has multiple channels, we suggest separating
// the different color channels so that each color is stored contiguously
// instead of being interleaved. This will simplify your code.

// That is instead of RGBARGBARGBARGBA... we suggest transforming to three
// arrays (as in the previous homework we ignore the alpha channel again):
//  1) RRRRRRRR...
//  2) GGGGGGGG...
//  3) BBBBBBBB...
//
// The original layout is known an Array of Structures (AoS) whereas the
// format we are converting to is known as a Structure of Arrays (SoA).

// As a warm-up, we will ask you to write the kernel that performs this
// separation. You should then write the "meat" of the assignment,
// which is the kernel that performs the actual blur. We provide code that
// re-combines your blurred results for each color channel.

//****************************************************************************

// You must fill in the gaussian_blur kernel to perform the blurring of the
// inputChannel, using the array of weights, and put the result in the outputChannel.

// Here is an example of computing a blur, using a weighted average, for a single
// pixel in a small image.
//
// Array of weights:
//
//  0.0  0.2  0.0
//  0.2  0.2  0.2
//  0.0  0.2  0.0
//
// Image (note that we align the array of weights to the center of the box):
//
//    1  2  5  2  0  3
//       -------
//    3 |2  5  1| 6  0       0.0*2 + 0.2*5 + 0.0*1 +
//      |       |
//    4 |3  6  2| 1  4   ->  0.2*3 + 0.2*6 + 0.2*2 +   ->  3.2
//      |       |
//    0 |4  0  3| 4  2       0.0*4 + 0.2*0 + 0.0*3
//       -------
//    9  6  5  0  3  9
//
//         (1)                         (2)                 (3)
//
// A good starting place is to map each thread to a pixel as you have before.
// Then every thread can perform steps 2 and 3 in the diagram above
// completely independently of one another.

// Note that the array of weights is square, so its height is the same as its width.
// We refer to the array of weights as a filter, and we refer to its width with the
// variable filterWidth.

//****************************************************************************

// Your homework submission will be evaluated based on correctness and speed.
// We test each pixel against a reference solution. If any pixel differs by
// more than some small threshold value, the system will tell you that your
// solution is incorrect, and it will let you try again.

// Once you have gotten that working correctly, then you can think about using
// shared memory and having the threads cooperate to achieve better performance.

//****************************************************************************

// Also note that we've supplied a helpful debugging function called checkCudaErrors.
// You should wrap your allocation and copying statements like we've done in the
// code we're supplying you. Here is an example of the unsafe way to allocate
// memory on the GPU:
//
// hipMalloc(&d_red, sizeof(unsigned char) * numRows * numCols);
//
// Here is an example of the safe way to do the same thing:
//
// checkCudaErrors(hipMalloc(&d_red, sizeof(unsigned char) * numRows * numCols));
//
// Writing code the safe way requires slightly more typing, but is very helpful for
// catching mistakes. If you write code the unsafe way and you make a mistake, then
// any subsequent kernels won't compute anything, and it will be hard to figure out
// why. Writing code the safe way will inform you as soon as you make a mistake.

// Finally, remember to free the memory you allocate at the end of the function.

//****************************************************************************

#include "utils.h"
#define BLOCK_WIDTH 16
#define FILTER_WIDTH 9                             // HW2.cpp 硬编码
#define FILTER_RADIUS (FILTER_WIDTH / 2)           // -> 4
#define TILE_DIM (BLOCK_WIDTH + 2 * FILTER_RADIUS) // -> 16 + 8 = 24

__global__ void gaussian_blur_shared(const unsigned char *const inputChannel,
                                     unsigned char *const outputChannel,
                                     int numRows, int numCols,
                                     const float *const filter, const int filterWidth)
{
  // 声明共享内存
  __shared__ unsigned char tile[TILE_DIM][TILE_DIM];

  // 线程在块内的局部坐标
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;

  // 计算块级共享的 tile 在全局内存中的起始坐标
  const int tile_start_x = blockIdx.x * BLOCK_WIDTH - FILTER_RADIUS;
  const int tile_start_y = blockIdx.y * BLOCK_WIDTH - FILTER_RADIUS;

  // --- 1. 从全局内存协同加载数据到共享内存 tile ---
  for (int i = 0; i < TILE_DIM * TILE_DIM; i += blockDim.x * blockDim.y)
  {
    int idx = i + ty * blockDim.x + tx;
    if (idx < TILE_DIM * TILE_DIM)
    {
      int r = idx / TILE_DIM; // tile 内的行
      int c = idx % TILE_DIM; // tile 内的列

      // 计算要加载的全局内存坐标 (基于块的起始地址)
      int g_load_x = tile_start_x + c;
      int g_load_y = tile_start_y + r;

      // 边界钳位 (Clamping)
      if (g_load_x < 0)
        g_load_x = 0;
      if (g_load_x >= numCols)
        g_load_x = numCols - 1;
      if (g_load_y < 0)
        g_load_y = 0;
      if (g_load_y >= numRows)
        g_load_y = numRows - 1;

      tile[r][c] = inputChannel[g_load_y * numCols + g_load_x];
    }
  }

  // --- 2. 同步！确保所有线程都完成了加载 ---
  __syncthreads();

  // --- 3. 从共享内存进行计算 ---
  const int out_x = blockIdx.x * BLOCK_WIDTH + tx;
  const int out_y = blockIdx.y * BLOCK_WIDTH + ty;

  if (out_x < numCols && out_y < numRows)
  {
    float blurred_pix = 0.0f;
    for (int r = -FILTER_RADIUS; r <= FILTER_RADIUS; r++)
    {
      for (int c = -FILTER_RADIUS; c <= FILTER_RADIUS; c++)
      {
        // 读取共享内存中的数据，索引相对于线程在 tile 中的中心位置
        blurred_pix += tile[ty + FILTER_RADIUS + r][tx + FILTER_RADIUS + c] *
                       filter[(r + FILTER_RADIUS) * FILTER_WIDTH + (c + FILTER_RADIUS)];
      }
    }

    outputChannel[out_y * numCols + out_x] = (unsigned char)(blurred_pix);
    // outputChannel[out_y * numCols + out_x] = (unsigned char)(blurred_pix + 0.5f);
  }
}

__global__ void gaussian_blur(const unsigned char *const inputChannel,
                              unsigned char *const outputChannel,
                              int numRows, int numCols,
                              const float *const filter, const int filterWidth)
{
  // TODO
  // 使用全局内存
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x >= numCols || y >= numRows)
  {
    return;
  }

  float blurred_pix = 0.0;

  int filterR = filterWidth / 2;

  for (int i = -filterR; i <= filterR; i++)
  {
    for (int j = -filterR; j <= filterR; j++)
    {
      int neighbor_x = x + i;
      int neighbor_y = y + j;

      if (neighbor_x < 0)
        neighbor_x = 0;
      if (neighbor_y < 0)
        neighbor_y = 0;
      if (neighbor_x >= numCols)
        neighbor_x = numCols - 1;
      if (neighbor_y >= numRows)
        neighbor_y = numRows - 1;

      int idx_pix = neighbor_y * numCols + neighbor_x;
      int filter_idx = (i + filterR) * filterWidth + (j + filterR);

      blurred_pix += (float)inputChannel[idx_pix] * filter[filter_idx];
    }
  }
  int output_idx = y * numCols + x;
  outputChannel[output_idx] = (unsigned char)(blurred_pix); // 默认直接截断
  // outputChannel[output_idx] = (unsigned char)(blurred_pix + 0.5f); // 四舍五入
  // NOTE: Be sure to compute any intermediate results in floating point
  // before storing the final result as unsigned char.

  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  //
  // if ( absolute_image_position_x >= numCols ||
  //      absolute_image_position_y >= numRows )
  // {
  //     return;
  // }

  // NOTE: If a thread's absolute position 2D position is within the image, but some of
  // its neighbors are outside the image, then you will need to be extra careful. Instead
  // of trying to read such a neighbor value from GPU memory (which won't work because
  // the value is out of bounds), you should explicitly clamp the neighbor values you read
  // to be within the bounds of the image. If this is not clear to you, then please refer
  // to sequential reference solution for the exact clamping semantics you should follow.
}

// This kernel takes in an image represented as a uchar4 and splits
// it into three images consisting of only one color channel each
__global__ void separateChannels(const uchar4 *const inputImageRGBA,
                                 int numRows,
                                 int numCols,
                                 unsigned char *const redChannel,
                                 unsigned char *const greenChannel,
                                 unsigned char *const blueChannel)
{
  // TODO
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x >= numCols || y >= numRows)
  {
    return;
  }

  int idx_1D = y * numCols + x;

  uchar4 RGB = inputImageRGBA[idx_1D];

  redChannel[idx_1D] = RGB.x;
  greenChannel[idx_1D] = RGB.y;
  blueChannel[idx_1D] = RGB.z;

  // NOTE: Be careful not to try to access memory that is outside the bounds of
  // the image. You'll want code that performs the following check before accessing
  // GPU memory:
  //
  // if ( absolute_image_position_x >= numCols ||
  //      absolute_image_position_y >= numRows )
  // {
  //     return;
  // }
}

// This kernel takes in three color channels and recombines them
// into one image.  The alpha channel is set to 255 to represent
// that this image has no transparency.
__global__ void recombineChannels(const unsigned char *const redChannel,
                                  const unsigned char *const greenChannel,
                                  const unsigned char *const blueChannel,
                                  uchar4 *const outputImageRGBA,
                                  int numRows,
                                  int numCols)
{
  const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                       blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  // make sure we don't try and access memory outside the image
  // by having any threads mapped there return early
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue = blueChannel[thread_1D_pos];

  // Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float *const h_filter, const size_t filterWidth)
{

  // allocate memory for the three different channels
  // original
  checkCudaErrors(hipMalloc(&d_red, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue, sizeof(unsigned char) * numRowsImage * numColsImage));

  // TODO:
  // Allocate memory for the filter on the GPU
  // Use the pointer d_filter that we have already declared for you
  // You need to allocate memory for the filter with hipMalloc
  // be sure to use checkCudaErrors like the above examples to
  // be able to tell if anything goes wrong
  // IMPORTANT: Notice that we pass a pointer to a pointer to hipMalloc

  checkCudaErrors(hipMalloc((void **)&d_filter, sizeof(float) * filterWidth * filterWidth));

  // TODO:
  // Copy the filter on the host (h_filter) to the memory you just allocated
  // on the GPU.  hipMemcpy(dst, src, numBytes, hipMemcpyHostToDevice);
  // Remember to use checkCudaErrors!

  checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));
}

void your_gaussian_blur(const uchar4 *const h_inputImageRGBA, uchar4 *const d_inputImageRGBA,
                        uchar4 *const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redBlurred,
                        unsigned char *d_greenBlurred,
                        unsigned char *d_blueBlurred,
                        const int filterWidth)
{
  // TODO: Set reasonable block size (i.e., number of threads per block)
  const dim3 blockSize(BLOCK_WIDTH, BLOCK_WIDTH, 1);

  // TODO:
  // Compute correct grid size (i.e., number of blocks per kernel launch)
  // from the image size and and block size.
  const dim3 gridSize((numCols + blockSize.x - 1) / blockSize.x,
                      (numRows + blockSize.y - 1) / blockSize.y,
                      1);

  // TODO: Launch a kernel for separating the RGBA image into different color channels

  separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA, numRows, numCols, d_red, d_green, d_blue);

  // Call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.

  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());

  // TODO: Call your convolution kernel here 3 times, once for each color channel.
  gaussian_blur_shared<<<gridSize, blockSize>>>(d_red, d_redBlurred, numRows, numCols, d_filter, filterWidth);
  gaussian_blur_shared<<<gridSize, blockSize>>>(d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);
  gaussian_blur_shared<<<gridSize, blockSize>>>(d_blue, d_blueBlurred, numRows, numCols, d_filter, filterWidth);

  // gaussian_blur<<<gridSize, blockSize>>>(d_red, d_redBlurred, numRows, numCols, d_filter, filterWidth);
  // gaussian_blur<<<gridSize, blockSize>>>(d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);
  // gaussian_blur<<<gridSize, blockSize>>>(d_blue, d_blueBlurred, numRows, numCols, d_filter, filterWidth);
  // Again, call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());

  // Now we recombine your results. We take care of launching this kernel for you.
  //
  // NOTE: This kernel launch depends on the gridSize and blockSize variables,
  // which you must set yourself.
  recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                             d_greenBlurred,
                                             d_blueBlurred,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());
}

// Free all the memory that we allocated
// TODO: make sure you free any arrays that you allocated
void cleanup()
{
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
  checkCudaErrors(hipFree(d_filter));
}
